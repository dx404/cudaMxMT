/**
 * Author: Duo Zhao
 * This part is the
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "seqMatrix.h"
#include "cuBLAS_MxMT.cuh"
#include "cudaGFlopTimer.cuh"

void cuBLAS_MxMT_device(float *d_r, float *d_m, int d){
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha = 1.0f, beta = 1.0f;
	// calling cuda buid-in library to executing computation
	hipblasSgemm(handle,
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			d, d ,d,
			&alpha,
			d_m, d,
			d_m, d,
			&beta,
			d_r, d);
}

float cuBLAS_MxMT_host(float *h_r, float *h_m, int d){
	cudaGFlopTimer *tr = new cudaGFlopTimer();

	float *d_m, *d_r;
	hipMalloc((void **) &d_m, d*d*sizeof(float));
	hipMalloc((void **) &d_r, d*d*sizeof(float));
	hipMemcpy(d_m, h_m, d*d*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, h_r, d*d*sizeof(float), hipMemcpyHostToDevice);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha = 1.0f, beta = 1.0f;

	// calling cuda buid-in library to executing computation
	tr->start();
	hipblasSgemm(handle,
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			d, d ,d,
			&alpha,
			d_m, d,
			d_m, d,
			&beta,
			d_r, d);
	tr->stop();

	hipMemcpy(h_r, d_r, d*d*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(d_m);
	hipFree(d_r);
	float Gflops = tr->getGFlops(d);
	delete tr;
	return Gflops;
}

