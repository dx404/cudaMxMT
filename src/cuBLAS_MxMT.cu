/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "seqMatrix.h"
#include "cuBLAS_MxMT.cuh"

float cuBLAS_MxMT(float *h_m, float *h_r, int d){
	float time;

	float *d_m, *d_r;
	hipMalloc((void **) &d_m, d*d*sizeof(float));
	hipMalloc((void **) &d_r, d*d*sizeof(float));
	hipMemcpy(d_m, h_m, d*d*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, h_r, d*d*sizeof(float), hipMemcpyHostToDevice);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha = 1.0f, beta = 1.0f;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	hipblasSgemm(handle,
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			d, d ,d,
			&alpha,
			d_m, d,
			d_m, d,
			&beta,
			d_r, d);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(h_r, d_r, d*d*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(d_m);
	hipFree(d_r);


	float gFLOPS =   ( 2.0e-6 * d * d * d) /(time);

	return gFLOPS;
}
