#include "cudaGFlopTimer.cuh"

Hi::Hi(){
  x = 5;
}

cudaGFlopTimer::cudaGFlopTimer(){
	hipEventCreate(&s);
	hipEventCreate(&t);
}

void cudaGFlopTimer::start(){
	hipEventRecord(s);
}

void cudaGFlopTimer::stop(){
	hipEventRecord(t);
	hipEventSynchronize(t);
	hipEventElapsedTime(&time, s, t);
}

float cudaGFlopTimer::getElapsedTime(){
	return time;
}

float cudaGFlopTimer::getGFlops(float d){
	return (2.0e-6 * d * d * d) /(time);
}



