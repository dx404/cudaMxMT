#include <iostream>
#include "cudaGFlopTimer.cuh"
using namespace std;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

float cuBLAS_MxMT(float *h_m, float *h_r, int d){
	cudaGFlopTimer *tr = new cudaGFlopTimer();

	float *d_m, *d_r;
	hipMalloc((void **) &d_m, d*d*sizeof(float));
	hipMalloc((void **) &d_r, d*d*sizeof(float));
	hipMemcpy(d_m, h_m, d*d*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, h_r, d*d*sizeof(float), hipMemcpyHostToDevice);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float alpha = 1.0f, beta = 1.0f;

	// calling cuda buid-in library to executing computation
	tr->start();
	hipblasSgemm(handle,
			HIPBLAS_OP_T, HIPBLAS_OP_N,
			d, d ,d,
			&alpha,
			d_m, d,
			d_m, d,
			&beta,
			d_r, d);
	//timer->stop();

	hipMemcpy(h_r, d_r, d*d*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(d_m);
	hipFree(d_r);
	float Gflops = 0; //= timer->getGFlops(d);
	return Gflops;
}




int main(int argc, char *argv[]){
  Hi *hi = new Hi();
  cudaGFlopTimer *cgt = new cudaGFlopTimer();
  cgt->start();
  cgt->stop();
  cgt->getElapsedTime();
  cgt->getGFlops(100);
  hi->x = 2;
  delete hi;
  delete cgt;
  return 0;
}
