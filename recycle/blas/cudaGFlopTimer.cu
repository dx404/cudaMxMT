#include <hip/hip_runtime.h>
#include "cudaGFlopTimer.cuh"

cudaGFlopTimer::cudaGFlopTimer(){
	hipEventCreate(&s);
	hipEventCreate(&t);
}

void cudaGFlopTimer::start(){
	hipEventRecord(s);
}

inline void cudaGFlopTimer::stop(){
	hipEventRecord(t);
	hipEventSynchronize(t);
	hipEventElapsedTime(&time, s, t);
}

inline float cudaGFlopTimer::getElapsedTime(){
	return time;
}

inline float cudaGFlopTimer::getGFlops(float d){
	return (2.0e-6 * d * d * d) /(time);
}

int main(){
	cudaGFlopTimer *timer = new cudaGFlopTimer();
	timer->start();
}

